/****************************************************************************
 * Mini-application to test performance optimizations for a weighted track
 * filter and vertex fitter.
 *
 * Authors: Joshua Shterenberg, Daniel Wilson
 *
 * This mini-application represents and is derived from code in the
 * https://github.com/cms-sw/cmssw project, which is released with an Apache-2.0 license.
 ***************************************************************************/
//
//nvcc -arch sm_35 test_vertex_fitter_CUDA.cu -o test_vertex_fitter_CUDA
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <pthread.h>
#include <memory>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define NUM_VERTICES		10
#define NUM_TRACKS_PER_VERTEX	50
#define NUM_TRACKS		500 //NUM_VERTICES * NUM_TRACKS_PER_VERTEX
#define SAMPLE_NUM		12

struct track_soa_t {
  long int* ids;
  double* zs;
  double* weight;
  long int* vertex_ids;
  long int* cluster_ids;
};

__global__ void proc(double* tracks_zs, double* tracks_weight, long int* tracks_cluster_ids, double* z_vals) {
  // current parallel strat: every track gets its own thread, every block gets its own vertex
  //   (should be ok because 1024 max threads/block)
  // assuming the grid is 1d
  // THIS IS A BAD IDEA IN PRACTICE BECAUSE NUM_TRACKS_PER_VERTEX ISN'T CONSTANT

  const int i_track = blockIdx.x * blockDim.x + threadIdx.x; //track
  const int i_vertex = blockIdx.x; //vertex
  __shared__ unsigned cluster_track_count[NUM_VERTICES];
  __shared__ double cluster_track_mean[NUM_VERTICES];
  __shared__ double cluster_track_std[NUM_VERTICES];
  __shared__ double cluster_sum_of_weights[NUM_VERTICES];

  cluster_track_count[i_vertex] = 0;
  cluster_track_mean[i_vertex] = 0;
  cluster_track_std[i_vertex] = 0;
  cluster_sum_of_weights[i_vertex] = 0;

  __syncthreads(); //always sync write/read clusters

  long int cluster_id = tracks_cluster_ids[i_track];
  cluster_track_mean[cluster_id] += tracks_zs[i_track];
  cluster_track_count[cluster_id] += 1;

  __syncthreads();

  cluster_track_mean[i_vertex] /= cluster_track_count[i_vertex];

  __syncthreads();

  double diff = tracks_zs[i_track] - cluster_track_mean[cluster_id];
  cluster_track_std[cluster_id] += diff * diff;

  __syncthreads();

  cluster_track_std[i_vertex] = sqrt(cluster_track_std[i_vertex] / (cluster_track_count[i_vertex] - 1));

  __syncthreads();

  double xmstd;
  if (diff > cluster_track_std[cluster_id] * 3) {
    tracks_weight[i_track] = 0;
  } else {
    xmstd = (tracks_zs[i_track] - cluster_track_mean[cluster_id]) / cluster_track_std[cluster_id];
    tracks_weight[i_track] = exp(-0.5 * xmstd * xmstd) / (cluster_track_std[cluster_id] * sqrt(2 * M_PI));
  }
  cluster_sum_of_weights[cluster_id] += tracks_weight[i_track];

  __syncthreads();

  z_vals[cluster_id] += tracks_zs[i_track] * tracks_weight[i_track];

  __syncthreads();

  z_vals[i_vertex] /= cluster_sum_of_weights[i_vertex];



  /*
  for (i = 0; i < NUM_VERTICES; ++i) {
          cluster_track_count[i] = 0;
          cluster_track_mean[i] = 0;
          cluster_track_std[i] = 0;
          cluster_sum_of_weights[i] = 0;
  }
  for (i = 0; i < NUM_TRACKS; ++i) {
    const long int cluster_id = tracks_cluster_ids[i];
    cluster_track_mean[cluster_id] += tracks_zs[i];
    cluster_track_count[cluster_id] += 1;
  }
  for (i = 0; i < NUM_VERTICES; ++i) cluster_track_mean[i] /= cluster_track_count[i];
  for (i = 0; i < NUM_TRACKS; ++i) {
    const long int cluster_id = tracks_cluster_ids[i];
    const double diff = (tracks_zs[i] - cluster_track_mean[cluster_id]);
    cluster_track_std[cluster_id] += diff * diff;
  }
  for (i = 0; i < NUM_VERTICES; ++i) cluster_track_std[i] = sqrt(cluster_track_std[i] / (cluster_track_count[i] - 1));
  for (i = 0; i < NUM_TRACKS; ++i) {
    const long int cluster_id = tracks_cluster_ids[i];
    const double diff = (tracks_zs[i] - cluster_track_mean[cluster_id]);
    double xmstd;
    if (diff > cluster_track_std[cluster_id] * 3) {
      tracks_weight[i] = 0;
    } else {
      xmstd = ((tracks_zs[i] - cluster_track_mean[cluster_id]) / cluster_track_std[cluster_id]);
      tracks_weight[i] = exp(-0.5 * xmstd * xmstd) / (cluster_track_std[cluster_id] * sqrt(2 * M_PI));
    }
    cluster_sum_of_weights[cluster_id] += tracks_weight[i];
  }
  for (i = 0; i < NUM_TRACKS; ++i) {
    const long int cluster_id = tracks_cluster_ids[i];
    z_vals[cluster_id] += tracks_zs[i] * tracks_weight[i];
  }
  for (i = 0; i < NUM_VERTICES; ++i) z_vals[i] /= cluster_sum_of_weights[i];
  */
}

int main(int argc, char *argv[]) {
  //--------------------------------------------preproc
  int i, j;
  srand(time(NULL));

  double TRUE_Z_VALS[NUM_VERTICES], z_vals[NUM_VERTICES];
  for (i = 0; i < NUM_VERTICES; i++) {
    TRUE_Z_VALS[i] = -10 + (rand() * 20.0 / RAND_MAX);
    z_vals[i] = 0;
  }

  track_soa_t tracks;
  
  tracks.ids = (long int*) malloc(NUM_TRACKS * sizeof(long int));
  tracks.zs = (double*) malloc(NUM_TRACKS * sizeof(double));
  tracks.weight = (double*) malloc(NUM_TRACKS * sizeof(double));
  tracks.vertex_ids = (long int*) malloc(NUM_TRACKS * sizeof(long int));
  tracks.cluster_ids = (long int*) malloc(NUM_TRACKS * sizeof(long int));
  
  //tracks.zs = std::make_unique<double[]>(NUM_TRACKS);
  //tracks.weight = std::make_unique<double[]>(NUM_TRACKS);
  //tracks.vertex_ids = std::make_unique<long int[]>(NUM_TRACKS);
  //tracks.cluster_ids = std::make_unique<long int[]>(NUM_TRACKS);
  
  for (i = 0; i < NUM_TRACKS; i++) {
    tracks.ids[i] = i;
    tracks.vertex_ids[i] = i / NUM_TRACKS_PER_VERTEX;
    double track_pos = 0;
    for (j = 0; j < SAMPLE_NUM; j++) {
      track_pos += (double)rand() / RAND_MAX;
    }
    track_pos -= 6;
    tracks.zs[i] = (track_pos * 2 / (SAMPLE_NUM)) + TRUE_Z_VALS[i / NUM_TRACKS_PER_VERTEX];
    tracks.cluster_ids[i] = tracks.vertex_ids[i];
  }

  /////////////////////////CUDA////////////////////////
  CUDA_SAFE_CALL(hipSetDevice(0));
  double *tracks_zs_gpu, *tracks_weight_gpu, *z_vals_gpu;
  long int *tracks_cluster_ids_gpu;
  size_t track_size = NUM_TRACKS * sizeof(double), 
         vertex_size = NUM_VERTICES * sizeof(double),
         int_size = NUM_TRACKS * sizeof(long int);

  CUDA_SAFE_CALL(hipMalloc(&tracks_zs_gpu, track_size));
  CUDA_SAFE_CALL(hipMalloc(&tracks_weight_gpu, track_size));
  CUDA_SAFE_CALL(hipMalloc(&tracks_cluster_ids_gpu, int_size));
  CUDA_SAFE_CALL(hipMalloc(&z_vals_gpu, vertex_size));
  CUDA_SAFE_CALL(hipMemcpy(tracks_zs_gpu, tracks.zs, track_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(tracks_weight_gpu, tracks.weight, track_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(tracks_cluster_ids_gpu, tracks.cluster_ids, int_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(z_vals_gpu, z_vals, vertex_size, hipMemcpyHostToDevice));

  dim3 dimGrid(1);
  dim3 dimBlock(1);
  proc<<<dimGrid, dimBlock>>>
    (tracks_zs_gpu, tracks_weight_gpu, tracks_cluster_ids_gpu, z_vals);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipPeekAtLastError());

  CUDA_SAFE_CALL(hipMemcpy(z_vals, z_vals_gpu, vertex_size, hipMemcpyDeviceToHost));
  /////////////////////////CUDA////////////////////////

  //--------------------------------------------postproc
  double mean_square_error = 0;
  for (i = 0; i < NUM_VERTICES; i++) {
    double err = TRUE_Z_VALS[i] - z_vals[i];
    mean_square_error += err * err;
  }
  mean_square_error /= NUM_VERTICES;
  printf("Mean square error of z positions: %g\n", mean_square_error);

  return 0;
}
