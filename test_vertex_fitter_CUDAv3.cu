#include "hip/hip_runtime.h"
/****************************************************************************
 * Mini-application to test performance optimizations for a weighted track
 * filter and vertex fitter.
 *
 * Authors: Joshua Shterenberg, Daniel Wilson
 *
 * This mini-application represents and is derived from code in the
 * https://github.com/cms-sw/cmssw project, which is released with an Apache-2.0 license.
 ***************************************************************************/
//
//nvcc -arch sm_35 test_vertex_fitter_CUDAv3.cu -o test_vertex_fitter_CUDAv3
//

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <pthread.h>
#include <memory>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

#define PRINT_TIME		1
#define NUM_VERTICES		10
#define NUM_TRACKS_PER_VERTEX	50
#define NUM_TRACKS		500 //NUM_VERTICES * NUM_TRACKS_PER_VERTEX
#define SAMPLE_NUM		12

struct track_soa_t {
  long int* ids;
  double* zs;
  double* weight;
  long int* vertex_ids;
  long int* cluster_ids;
};

__global__ void proc(double* tracks_zs, double* tracks_weight, long int* tracks_cluster_ids, double* z_vals) {
  // current parallel strat: every track gets its own thread, block/vertex

  const int i_track = threadIdx.x; //track
  const int i_vertex = blockDim.x; //vertex
  __shared__ unsigned cluster_track_count = 0;
  __shared__ double cluster_track_mean = 0;
  __shared__ double cluster_track_std = 0;
  __shared__ double cluster_sum_of_weights = 0;

  __syncthreads(); //always sync write/read clusters

  long int cluster_id = tracks_cluster_ids[i_track];
  atomicAdd(&cluster_track_mean, tracks_zs[i_track]);
  atomicAdd(&cluster_track_count, 1);
  
  __syncthreads();

  cluster_track_mean /= cluster_track_count;

  __syncthreads();

  double diff = -1.0 * abs(tracks_zs[i_track] - cluster_track_mean);
  atomicAdd(&cluster_track_std, diff*diff);

  __syncthreads();

  cluster_track_std = sqrt(cluster_track_std / (cluster_track_count - 1));

  __syncthreads();

  if (diff <= cluster_track_std * 3) {
    double xmstd = (tracks_zs[i_track] - cluster_track_mean) / cluster_track_std;
    tracks_weight[i_track] = exp(-0.5 * xmstd * xmstd) / (cluster_track_std * sqrt(2 * M_PI));
  } else tracks_weight[i_track] = 0;

  atomicAdd(&cluster_sum_of_weights, tracks_weight[i_track]);

  __syncthreads();
 
  atomicAdd(&z_vals[i_vertex], tracks_zs[i_track] * tracks_weight[i_track]);

  __syncthreads();

  z_vals[i_vertex] /= cluster_sum_of_weights;

}

int main(int argc, char *argv[]) {

  hipEvent_t start, stop;
  float elapsed;

  //--------------------------------------------preproc
  int i, j;
  srand(time(NULL));

  double TRUE_Z_VALS[NUM_VERTICES], z_vals[NUM_VERTICES];
  for (i = 0; i < NUM_VERTICES; i++) {
    TRUE_Z_VALS[i] = -10 + (rand() * 20.0 / RAND_MAX);
    z_vals[i] = 0;
  }

  track_soa_t tracks;
  
  tracks.ids = (long int*) malloc(NUM_TRACKS * sizeof(long int));
  tracks.zs = (double*) malloc(NUM_TRACKS * sizeof(double));
  tracks.weight = (double*) malloc(NUM_TRACKS * sizeof(double));
  tracks.vertex_ids = (long int*) malloc(NUM_TRACKS * sizeof(long int));
  tracks.cluster_ids = (long int*) malloc(NUM_TRACKS * sizeof(long int));
  
  for (i = 0; i < NUM_TRACKS; i++) {
    tracks.ids[i] = i;
    tracks.vertex_ids[i] = i / NUM_TRACKS_PER_VERTEX;
    double track_pos = 0;
    for (j = 0; j < SAMPLE_NUM; j++) {
      track_pos += (double)rand() / RAND_MAX;
    }
    track_pos -= 6;
    tracks.zs[i] = (track_pos * 2 / (SAMPLE_NUM)) + TRUE_Z_VALS[i / NUM_TRACKS_PER_VERTEX];
    tracks.cluster_ids[i] = tracks.vertex_ids[i];
  }

  /////////////////////////CUDA////////////////////////
  CUDA_SAFE_CALL(hipSetDevice(0));
  double *tracks_zs_gpu, *tracks_weight_gpu, *z_vals_gpu;
  long int *tracks_cluster_ids_gpu;
  size_t track_size = NUM_TRACKS * sizeof(double), 
         vertex_size = NUM_VERTICES * sizeof(double),
         int_size = NUM_TRACKS * sizeof(long int);

  CUDA_SAFE_CALL(hipMalloc(&tracks_zs_gpu, track_size));
  CUDA_SAFE_CALL(hipMalloc(&tracks_weight_gpu, track_size));
  CUDA_SAFE_CALL(hipMalloc(&tracks_cluster_ids_gpu, int_size));
  CUDA_SAFE_CALL(hipMalloc(&z_vals_gpu, vertex_size));
  CUDA_SAFE_CALL(hipMemcpy(tracks_zs_gpu, tracks.zs, track_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(tracks_weight_gpu, tracks.weight, track_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(tracks_cluster_ids_gpu, tracks.cluster_ids, int_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(z_vals_gpu, z_vals, vertex_size, hipMemcpyHostToDevice));

  dim3 dimGrid(NUM_VERTICES);
  dim3 dimBlock(NUM_TRACKS_PER_VERTEX);

#if PRINT_TIME
  // Create the cuda events
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Record event on the default stream
  hipEventRecord(start, 0);
#endif
  proc<<<dimGrid, dimBlock>>>
    (tracks_zs_gpu, tracks_weight_gpu, tracks_cluster_ids_gpu, z_vals_gpu);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipPeekAtLastError());
#if PRINT_TIME
  // Stop and destroy the timer
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif
  CUDA_SAFE_CALL(hipMemcpy(z_vals, z_vals_gpu, vertex_size, hipMemcpyDeviceToHost));
  /////////////////////////CUDA////////////////////////

  //--------------------------------------------postproc

  double mean_square_error = 0;
  for (i = 0; i < NUM_VERTICES; i++) {
    double err = TRUE_Z_VALS[i] - z_vals[i];
    mean_square_error += err * err;
  }
  mean_square_error /= NUM_VERTICES;
  printf("GPU implementation mean square error of z positions: %g\n", mean_square_error);
  printf("GPU implementation time (s): %f (msec)\n", elapsed);

  return 0;
}
